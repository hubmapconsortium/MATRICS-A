#include "hip/hip_runtime.h"
/*
 *  _reg_resampling_gpu.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_RESAMPLING_GPU_CU
#define _REG_RESAMPLING_GPU_CU

#include "_reg_resampling_gpu.h"
#include "_reg_resampling_kernels.cu"

/* *************************************************************** */
/* *************************************************************** */
void reg_resampleSourceImage_gpu(nifti_image *sourceImage,
                                float **resultImageArray_d,
                                hipArray **sourceImageArray_d,
                                float4 **positionFieldImageArray_d,
                                int **mask_d,
                                int activeVoxelNumber,
                                float sourceBGValue)
{
    int3 sourceDim = make_int3(sourceImage->nx, sourceImage->ny, sourceImage->nz);

    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_SourceDim),&sourceDim,sizeof(int3)))
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_PaddingValue),&sourceBGValue,sizeof(float)))
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber),&activeVoxelNumber,sizeof(int)))

    //Bind source image array to a 3D texture
    sourceTexture.normalized = true;
    sourceTexture.filterMode = hipFilterModeLinear;
    sourceTexture.addressMode[0] = hipAddressModeWrap;
    sourceTexture.addressMode[1] = hipAddressModeWrap;
    sourceTexture.addressMode[2] = hipAddressModeWrap;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    NR_CUDA_SAFE_CALL(hipBindTextureToArray(sourceTexture, *sourceImageArray_d, channelDesc))

    //Bind positionField to texture
    NR_CUDA_SAFE_CALL(hipBindTexture(0, positionFieldTexture, *positionFieldImageArray_d, activeVoxelNumber*sizeof(float4)))

    //Bind positionField to texture
    NR_CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, *mask_d, activeVoxelNumber*sizeof(int)))

    // Bind the real to voxel matrix to texture
    mat44 *sourceMatrix;
    if(sourceImage->sform_code>0)
        sourceMatrix=&(sourceImage->sto_ijk);
    else sourceMatrix=&(sourceImage->qto_ijk);
    float4 *sourceRealToVoxel_h;NR_CUDA_SAFE_CALL(hipHostMalloc(&sourceRealToVoxel_h, 3*sizeof(float4)))
    float4 *sourceRealToVoxel_d;
    NR_CUDA_SAFE_CALL(hipMalloc(&sourceRealToVoxel_d, 3*sizeof(float4)))
    for(int i=0; i<3; i++){
        sourceRealToVoxel_h[i].x=sourceMatrix->m[i][0];
        sourceRealToVoxel_h[i].y=sourceMatrix->m[i][1];
        sourceRealToVoxel_h[i].z=sourceMatrix->m[i][2];
        sourceRealToVoxel_h[i].w=sourceMatrix->m[i][3];
    }
    NR_CUDA_SAFE_CALL(hipMemcpy(sourceRealToVoxel_d, sourceRealToVoxel_h, 3*sizeof(float4), hipMemcpyHostToDevice))
    NR_CUDA_SAFE_CALL(hipHostFree((void *)sourceRealToVoxel_h))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, sourceMatrixTexture, sourceRealToVoxel_d, 3*sizeof(float4)))

    const unsigned int Grid_reg_resampleSourceImage = (unsigned int)ceil(sqrtf((float)activeVoxelNumber/(float)Block_reg_resampleSourceImage));
    dim3 B1(Block_reg_resampleSourceImage,1,1);
    dim3 G1(Grid_reg_resampleSourceImage,Grid_reg_resampleSourceImage,1);
    reg_resampleSourceImage_kernel <<< G1, B1 >>> (*resultImageArray_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)

    NR_CUDA_SAFE_CALL(hipUnbindTexture(sourceTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(positionFieldTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(maskTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(sourceMatrixTexture))

    hipFree(sourceRealToVoxel_d);
}
/* *************************************************************** */
/* *************************************************************** */
void reg_getSourceImageGradient_gpu(nifti_image *sourceImage,
                                    hipArray **sourceImageArray_d,
                                    float4 **positionFieldImageArray_d,
                                    float4 **resultGradientArray_d,
                                    int activeVoxelNumber)
{
    int3 sourceDim = make_int3(sourceImage->nx, sourceImage->ny, sourceImage->nz);

    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_SourceDim), &sourceDim, sizeof(int3)))
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber), &activeVoxelNumber, sizeof(int)))

    //Bind source image array to a 3D texture
    sourceTexture.normalized = true;
    sourceTexture.filterMode = hipFilterModeLinear;
    sourceTexture.addressMode[0] = hipAddressModeWrap;
    sourceTexture.addressMode[1] = hipAddressModeWrap;
    sourceTexture.addressMode[2] = hipAddressModeWrap;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    NR_CUDA_SAFE_CALL(hipBindTextureToArray(sourceTexture, *sourceImageArray_d, channelDesc))

    //Bind positionField to texture
    NR_CUDA_SAFE_CALL(hipBindTexture(0, positionFieldTexture, *positionFieldImageArray_d, activeVoxelNumber*sizeof(float4)))

    // Bind the real to voxel matrix to texture
    mat44 *sourceMatrix;
    if(sourceImage->sform_code>0)
        sourceMatrix=&(sourceImage->sto_ijk);
    else sourceMatrix=&(sourceImage->qto_ijk);
    float4 *sourceRealToVoxel_h;NR_CUDA_SAFE_CALL(hipHostMalloc(&sourceRealToVoxel_h, 3*sizeof(float4)))
    float4 *sourceRealToVoxel_d;
    NR_CUDA_SAFE_CALL(hipMalloc(&sourceRealToVoxel_d, 3*sizeof(float4)))
    for(int i=0; i<3; i++){
        sourceRealToVoxel_h[i].x=sourceMatrix->m[i][0];
        sourceRealToVoxel_h[i].y=sourceMatrix->m[i][1];
        sourceRealToVoxel_h[i].z=sourceMatrix->m[i][2];
        sourceRealToVoxel_h[i].w=sourceMatrix->m[i][3];
    }
    NR_CUDA_SAFE_CALL(hipMemcpy(sourceRealToVoxel_d, sourceRealToVoxel_h, 3*sizeof(float4), hipMemcpyHostToDevice))
    NR_CUDA_SAFE_CALL(hipHostFree((void *)sourceRealToVoxel_h))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, sourceMatrixTexture, sourceRealToVoxel_d, 3*sizeof(float4)))

    const unsigned int Grid_reg_getSourceImageGradient = (unsigned int)ceil(sqrtf((float)activeVoxelNumber/(float)Block_reg_getSourceImageGradient));
    dim3 B1(Block_reg_getSourceImageGradient,1,1);
    dim3 G1(Grid_reg_getSourceImageGradient,Grid_reg_getSourceImageGradient,1);
    reg_getSourceImageGradient_kernel <<< G1, B1 >>> (*resultGradientArray_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(sourceTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(positionFieldTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(sourceMatrixTexture))

    hipFree(sourceRealToVoxel_d);
}
/* *************************************************************** */
/* *************************************************************** */

#endif
